#include "hip/hip_runtime.h"
#include "test_case_cuda.h"
#include "csr_matrix.h"
#include <iostream>
#include <omp.h>
#include <cstdio>
#include "gemm.h"
#include <algorithm>
#include "matrix_utils.h"
#include "spmm_cuda_ref.h"
#include "spmm_ref.h"
#include "spmm_cusparse.h"
#include "spmm_cuda_opt.h"
#include <chrono>
#include <hipsparse.h>


#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)


void test_spmm_cuda(const int m, const int n, const int k, const int test_time, const double sparsity) {
    // Host内存分配
    float* A = (float*)malloc(m * k * sizeof(float));
    float* B = (float*)malloc(k * n * sizeof(float));
    float* C = (float*)calloc(m * n, sizeof(float));
    float* C_gpu = (float*)calloc(m * n, sizeof(float));
    
    // 生成测试数据
    Gen_Matrix_sparsity(A, m, k, sparsity);
    Gen_Matrix(B, k, n);
    CSRMatrix<float>* csr_matrix = dense_to_csr(A, m, k);
    
    // 参考结果B
    spmm_cpu_ref(csr_matrix->row_ptr, csr_matrix->col_indices, csr_matrix->values, B, C, m, n,k);

    // Device内存分配
    int *d_ptr, *d_idx;
    float *d_val, *d_vin, *d_vout;
    hipMalloc(&d_ptr, (m + 1) * sizeof(int));
    hipMalloc(&d_idx, csr_matrix->nnz * sizeof(int));
    hipMalloc(&d_val, csr_matrix->nnz * sizeof(float));
    hipMalloc(&d_vin, k * n * sizeof(float));
    hipMalloc(&d_vout, m * n * sizeof(float));
    // Host to Device
    hipMemcpy(d_ptr, csr_matrix->row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_idx, csr_matrix->col_indices, csr_matrix->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, csr_matrix->values, csr_matrix->nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vin, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    // 性能测试
    float min_time = 1e6;

    hipEvent_t start, stop;
    for(int i = 0; i < test_time; i++) {
        hipMemset(d_vout, 0, m * n * sizeof(float));
        // hipDeviceSynchronize();
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);
        spmm_cuda_opt(d_ptr, d_idx, d_val, d_vin, d_vout, m, n,k);
        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time=0;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
        min_time=min(elapsed_time,min_time);
    }
    // Device to Host
    hipMemcpy(C_gpu, d_vout, m * n * sizeof(float), hipMemcpyDeviceToHost);
    // 验证结果
    float max_diff = max_diff_twoMatrix(C_gpu, C, m, n);
    bool is_correct = (max_diff < 1e-3);
    
    std::cout << "CUDA SpMM COST TIME: " << min_time << " ms " ;
    double gflops=(2.0*csr_matrix->nnz*n*1e-9)/(min_time/1000);
    std::cout << "CUDA SpMM GFLOPS: " << gflops << std::endl;
    std::cout << (is_correct ? "correct √" : "false !!")<< " max diff: " << max_diff << "\n";
    // 清理内存
    free(A); free(B); free(C); free(C_gpu);
    hipFree(d_ptr); hipFree(d_idx); hipFree(d_val); hipFree(d_vin); hipFree(d_vout);
    free_csr_matrix(csr_matrix);
}


void test_spmm_cusparse(const int m, const int n, const int k, const int test_time, const double sparsity) {
    // Host内存分配
    float* A = (float*)malloc(m * k * sizeof(float));
    float* B = (float*)malloc(k * n * sizeof(float));
    float* C_gpu = (float*)calloc(m * n, sizeof(float));
    
    // 生成测试数据
    Gen_Matrix_sparsity(A, m, k, sparsity);
    Gen_Matrix(B, k, n);
    CSRMatrix<float>* csr_matrix = dense_to_csr(A, m, k);
    // Device内存分配
    int *d_ptr, *d_idx;
    float *d_val, *d_vin, *d_vout;
    hipMalloc(&d_ptr, (m + 1) * sizeof(int));
    hipMalloc(&d_idx, csr_matrix->nnz * sizeof(int));
    hipMalloc(&d_val, csr_matrix->nnz * sizeof(float));
    hipMalloc(&d_vin, k * n * sizeof(float));
    hipMalloc(&d_vout, m * n * sizeof(float));
    // Host to Device
    hipMemcpy(d_ptr, csr_matrix->row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_idx, csr_matrix->col_indices, csr_matrix->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, csr_matrix->values, csr_matrix->nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vin, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    // 性能测试
    float min_time = 1e6;
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    
    // 创建稀疏矩阵描述符（CSR格式）
    hipsparseSpMatDescr_t matA;
    hipsparseCreateCsr(&matA, m, k, csr_matrix->nnz, d_ptr, d_idx, d_val,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // 创建稠密矩阵描述符（行优先存储）
    hipsparseDnMatDescr_t matB;
    hipsparseCreateDnMat(&matB, k, n, n, d_vin, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    // 设置SpMM参数
    float alpha = 1.0f;
    float beta = 0.0;  
    // 计算所需缓冲区大小
    hipsparseDnMatDescr_t matC;
    hipsparseCreateDnMat(&matC, m, n, n, d_vout, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    size_t bufferSize = 0;
    hipsparseSpMM_bufferSize(handle, 
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, matB, &beta, matC,
                            HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    void *dBuffer = nullptr;
    hipMalloc(&dBuffer, bufferSize);

    // 分配缓冲区
   
    hipEvent_t start, stop;
    for(int i = 0; i < test_time; i++) {
        hipDeviceSynchronize();
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        hipsparseSpMM(handle, 
                 HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha, matA, matB, &beta, matC,
                 HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);


        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time=0;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
        min_time=min(elapsed_time,min_time);
    }
    // Device to Host
    hipMemcpy(C_gpu, d_vout, m * n * sizeof(float), hipMemcpyDeviceToHost);
  
    std::cout << "CUSPARSE COST TIME: " << min_time << " ms ";
    double gflops=(2.0*csr_matrix->nnz*n*1e-9)/(min_time/1000);
    std::cout << " CUSPARSE GFLOPS: " << gflops << std::endl;
    // 清理内存
    free(A); free(B); free(C_gpu);
    hipFree(d_ptr); hipFree(d_idx); hipFree(d_val); hipFree(d_vin); hipFree(d_vout);
    free_csr_matrix(csr_matrix);
    
    hipFree(dBuffer);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(handle);
}