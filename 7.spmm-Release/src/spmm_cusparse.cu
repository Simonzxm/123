#include <hipsparse.h>
#include <spmm_cusparse.h>
#include <iostream>
void spmm_cusparse(int *d_ptr, int *d_idx, float *d_val, float *d_vin, float *d_vout, int m, int n, int k,int nnz)
{
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    
    // 创建稀疏矩阵描述符（CSR格式）
    hipsparseSpMatDescr_t matA;
    hipsparseCreateCsr(&matA, m, k, nnz, d_ptr, d_idx, d_val,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // 创建稠密矩阵描述符（行优先存储）
    hipsparseDnMatDescr_t matB;
    hipsparseCreateDnMat(&matB, k, n, n, d_vin, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseDnMatDescr_t matC;
    hipsparseCreateDnMat(&matC, m, n, n, d_vout, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    // 设置SpMM参数
    float alpha = 1.0f;
    float beta = 0.0;  

    // 计算所需缓冲区大小
    size_t bufferSize = 0;
    hipsparseSpMM_bufferSize(handle, 
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, matB, &beta, matC,
                            HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);

    // 分配缓冲区
    void *dBuffer = nullptr;
    hipMalloc(&dBuffer, bufferSize);

    // 执行SpMM
    hipsparseSpMM(handle, 
                 HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha, matA, matB, &beta, matC,
                 HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);

    // 清理资源
    hipFree(dBuffer);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matA);
    hipsparseDestroy(handle);
}